#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>

/*
 * A simple introduction to programming in CUDA. This program prints "Hello
 * World from GPU! from 10 CUDA threads running on the GPU.
 */




//this function rus in device  GPU

__global__ void helloFromGPU()
{
    printf("Hello World from GPU!\n");
}





// this is main runner of file on host CPU 

int main(int argc, char **argv)
{
    printf("Hello World from CPU!\n");

    helloFromGPU<<<1, 10>>>();
    CHECK(hipDeviceReset());
    return 0;
}

